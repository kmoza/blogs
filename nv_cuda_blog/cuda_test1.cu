
#include <hip/hip_runtime.h>
/* cuda kernel device code - cuda sample code
*computes the vector addition of A and B into C. The three vectors have
*the same number of elements*/
__global__ void vectorAdd(float *A, float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}
